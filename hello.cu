
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void){
    printf("Hello World.\n");
}

int main(void){
    printf("Hello World.\n");

    helloFromGPU <<<1 ,10>>> ();
    hipDeviceReset();

    return 0;
}